#include <stdio.h>

#include "cuda_globals.cuh"

#include "gl.cuh"
#include "const.cuh"
#include "globals.cuh"

static char* field = NULL;

static char* getFieldPtrAt(int x, int y)
{
    if (x > width || y > height) {
        return NULL;
    }
    return field + (x + (y * width));
}

static void unloadField()
{
    if (d_outfield) {
        hipFree(d_outfield);
        d_outfield = NULL;
    }
    if (d_field) {
        hipFree(d_field);
        d_field = NULL;
    }
    if (field) {
        free(field);
        field = NULL;
    }
}

static int loadFile(const char* fileName)
{
    unloadField();

    int fileWidth, fileHeight;

    FILE* fd = fopen(fileName, "rb");
    if (!fd) {
        return 1;
    }
    fscanf(fd, "%d %d", &fileWidth, &fileHeight);

    printf("Got file dimensions %d / %d\n", fileWidth, fileHeight);

    width = fileWidth + 2;
    height = fileHeight + 2;

    if (width % threadsPerBlock.x) {
        width += threadsPerBlock.x - (width % threadsPerBlock.x);
    }
    if (height % threadsPerBlock.y) {
        height += threadsPerBlock.y - (height % threadsPerBlock.y);
    }

    numBlocks.x = width / threadsPerBlock.x;
    numBlocks.y = height / threadsPerBlock.y;

    field = (char*)malloc(width*height);
    memset(field, CELL_EMPTY, width*height);

    char tmp;
    int x = 0;
    int y = 0;
    do {
        tmp = fgetc(fd);
        if (tmp == '\r' || tmp == '\n') {
            continue;
        }

        switch (tmp) {
            case FILE_CELL_CONDUCTOR:
                tmp = CELL_CONDUCTOR;
                break;
            case FILE_CELL_ELECTRON_HEAD:
                tmp = CELL_ELECTRON_HEAD;
                break;
            case FILE_CELL_ELECTRON_TAIL:
                tmp = CELL_ELECTRON_TAIL;
                break;
            default:
                tmp = CELL_EMPTY;
                break;
        }

        *getFieldPtrAt(x + 1, y + 1) = tmp;
        if (++x >= fileWidth) {
            y++;
            x = 0;
        }
    } while (!feof(fd));

    printf("File loaded: %s\n", fileName);
    fclose(fd);

    hipMalloc(&d_field, width*height);
    hipMalloc(&d_outfield, width*height);
    hipMemcpy(d_field, field, width*height, hipMemcpyHostToDevice);
    hipMemcpy(d_outfield, field, width*height, hipMemcpyHostToDevice);

    return 0;
}

int main(int argc, char** argv)
{
    if (loadFile(argv[1])) {
        unloadField();
        return 1;
    }

    if (initGL(&argc, argv)) {
        unloadField();
        return 1;
    }

    deinitGL();
    unloadField();
}
