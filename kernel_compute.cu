#include "hip/hip_runtime.h"
#include "kernel_shared.cuh"
#include "kernel_compute.cuh"
#include "cuda_globals.cuh"
#include "globals.cuh"
#include "const.cuh"

#include <stdio.h>
#include <chrono>

__device__ inline static char getFieldAtIsHead(int x, int y, int width, char* field)
{
    int offset = getFieldOffsetAt(x, y, width);
    if (offset < 0) {
        return 0;
    }
    return field[offset] == CELL_ELECTRON_HEAD;
}

__global__ void computeCell(int width, char* field, char* outfield)
{
    char tmp;
    int offset = getFieldOffsetAt(0, 0, width);

    switch (field[offset])
    {
        case CELL_ELECTRON_HEAD:
            outfield[offset] = CELL_ELECTRON_TAIL;
            break;
        case CELL_ELECTRON_TAIL:
            outfield[offset] = CELL_CONDUCTOR;
            break;
        case CELL_CONDUCTOR:
            tmp = 
                getFieldAtIsHead(-1, -1, width, field) +
                getFieldAtIsHead(-1, 0, width, field) +
                getFieldAtIsHead(-1, 1, width, field) +
                getFieldAtIsHead(0, -1, width, field) +
                getFieldAtIsHead(0, 1, width, field) +
                getFieldAtIsHead(1, -1, width, field) +
                getFieldAtIsHead(1, 0, width, field) +
                getFieldAtIsHead(1, 1, width, field);
            if (tmp == 1 || tmp == 2) {
                outfield[offset] = CELL_ELECTRON_HEAD;
                break;
            }
            outfield[offset] = CELL_CONDUCTOR;
            break;
    }
}

void runComputeCell(int iterations)
{
    for (int i = 0; i < iterations; i++) {
        computeCell<<<numBlocks, threadsPerBlock>>>(width, d_field, d_outfield);
        std::swap(d_outfield, d_field);
    }
    hipDeviceSynchronize();
}

static int timedIterations = 100;
void runComputeCellFor(float msTarget)
{
    auto t1 = std::chrono::high_resolution_clock::now();
    runComputeCell(timedIterations);
    auto t2 = std::chrono::high_resolution_clock::now();

    float msActual = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() / 1000.0f;
    timedIterations = timedIterations * (msTarget / msActual);
    if (timedIterations < 1) {
        timedIterations = 1;
    }
    printf("I: %d\n", timedIterations);
}
